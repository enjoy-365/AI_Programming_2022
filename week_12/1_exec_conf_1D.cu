
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void exec_conf(void) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	printf("tid = %d, threadIdx = (%d, %d, %d), blockIdx = (%d,%d,%d), blockDim = (%d,%d,%d), gridDim = (%d,%d,%d)\n",
			tid,
			threadIdx.x, threadIdx.y, threadIdx.z,
			blockIdx.x, blockIdx.y, blockIdx.z,
			blockDim.x, blockDim.y, blockDim.z,
			gridDim.x, gridDim.y, gridDim.z);
}

int main (void) {
	exec_conf<<<2,4>>>();
	hipDeviceSynchronize();
	return 0;
}

