// written by Jongsoo Kim
// Last modification: 2014-06-22
// compile options nvcc -Xptxas -v -arch sm_20 matmul_shared.cu 
// Intel(R) Xeon(R) CPU E5-2660 @ 2.20GHz and K40
// N= 1024, BLOCK_SIZE=32, 8.064341e+01 Gflops
// N= 2048, BLOCK_SIZE=32, 1.531437e+02 Gflops
// N= 4096, BLOCK_SIZE=32, 2.135609e+02 Gflops
// N= 8192, BLOCK_SIZE=32, 2.849405e+02 Gflops
// N=16384, BLOCK_SIZE=32, 3.266475e+02 Gflops

// Intel(R) Xeon(R) CPU           E5640  @ 2.67GHz
// N= 1024, BLOCK_SIZE=32, 1.223431e+02 Gflops
// N= 2048, BLOCK_SIZE=32, 1.546619e+02 Gflops
// N= 4096, BLOCK_SIZE=32, 1.656008e+02 Gflops 
// N= 8192, BLOCK_SIZE=32, 1.727456e+02 Gflops 


#include <hip/hip_runtime.h>
#include <stdio.h>  
#include <stdlib.h>  // for malloc
#include <time.h>    // for the clock() function

// size of a square matrix
const int N = 1024;

// number of threads in a block
const int BLOCK_SIZE = 32; 

__global__ void MatMul(const float * A, const float * B, float * C) {
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    float Cvalue = 0.0f;

    for (int tile=0; tile<N/BLOCK_SIZE; ++tile) {

        // loading of A and B matrices into shared memory
        As[ty][tx] = A[row * N + (tile*BLOCK_SIZE+tx)]; 
        Bs[ty][tx] = B[(tile*BLOCK_SIZE+ty)*N + col]; 
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) {
        Cvalue += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }
    C[row*N+col] = Cvalue;
}

int main (void) {

    float * A, * B, *C;    // arrays for host
    float * dA, * dB, *dC; // arrays for device 

    size_t size = N * N * sizeof(float);

    A = (float *) malloc(size);
    B = (float *) malloc(size);
    C = (float *) malloc(size);

    hipMalloc( (void**)&dA,size);
    hipMalloc( (void**)&dB,size);
    hipMalloc( (void**)&dC,size);

    // initialization of A and B matrices
    for (unsigned row=0; row<N; ++row)  
    for (unsigned col=0; col<N; ++col) { 
        A[row * N + col] = row + col + 1.0f; 
        B[row * N + col] = row + col + 1.0f; 
    }

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start,0 );

    // copy A and B from host to device
    hipMemcpy (dA, A, size, hipMemcpyHostToDevice);
    hipMemcpy (dB, B, size, hipMemcpyHostToDevice);

    // Imvoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);
    MatMul<<<dimGrid,dimBlock>>>(dA, dB, dC);

    // copy C from device  to host 
    hipMemcpy (C, dC, size, hipMemcpyDeviceToHost );

//    for (int row=0; row<N; ++row)  
//    for (int col=0; col<N; ++col) { 
//        printf("%f\n", C[row*N+col]);
//   }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float etime;
    hipEventElapsedTime( &etime, start, stop );
    etime = etime/1000.0;

    printf("elapsed time in seconds = %e\n",etime);
    double num_ops = 2.0f*N*N*N;
    printf("number of multiplications and additions  = %e\n",num_ops);
    printf("%e Gflops\n",num_ops/etime/1.e9);

    free(A); free(B); free(C);
    hipFree(dA); hipFree(dB); hipFree(dC);

    return 0;
}
