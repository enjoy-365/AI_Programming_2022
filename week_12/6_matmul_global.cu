// written by Jongsoo Kim
// Last modification: 2014-06-22
// compile options nvcc -Xptxas -v -arch sm_20 matmul_global.cu 
// Intel(R) Xeon(R) CPU E5-2660 @ 2.20GHz and K40
// N= 1024, BLOCK_SIZE=32, 7.482187e+01 Gflops
// N= 2048, BLOCK_SIZE=32, 1.030203e+02 Gflops
// N= 4096, BLOCK_SIZE=32, 1.208879e+02 Gflops
// N= 8192, BLOCK_SIZE=32, 1.278362e+02 Gflops
// N=16384, BLOCK_SIZE=32, 1.292009e+02 Gflops

// compile options nvcc -O3 -arch sm_20 matmul_global.cu         
// Intel(R) Xeon(R) CPU E5-2660 @ 2.20GHz and Tesla S2050 
// N= 1024, BLOCK_SIZE=32, 1.028392e+02 Gflops
// N= 2048, BLOCK_SIZE=32, 1.207798e+02 Gflops
// N= 4096, BLOCK_SIZE=32, 1.349969e+02 Gflops
// N= 8192, BLOCK_SIZE=32, 1.315237e+02 Gflops


#include <hip/hip_runtime.h>
#include <stdio.h>  
#include <stdlib.h>  // for malloc
#include <time.h>    // for the clock() function

// size of a square matrix
//const int N = 1024;
const int N = 8000;

// number of threads in a block
const int BLOCK_SIZE = 32; 

__global__ void MatMul(const float * A, const float * B, float * C) {
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float Cvalue = 0;
    for (int k = 0; k < N; ++k) {
        Cvalue += A[row * N + k] * B[k * N + col];
    }
    C[row*N+col] = Cvalue;
}

int main (void) {

    float * A, * B, *C;    // arrays for host
    float * dA, * dB, *dC; // arrays for device 

    size_t size = N * N * sizeof(float);

    A = (float *) malloc(size);
    B = (float *) malloc(size);
    C = (float *) malloc(size);

    hipMalloc( (void**)&dA,size);
    hipMalloc( (void**)&dB,size);
    hipMalloc( (void**)&dC,size);

    // initialization of A and B matrices
    for (unsigned row=0; row<N; ++row)  
    for (unsigned col=0; col<N; ++col) { 
        A[row * N + col] = row + col + 1.0f; 
        B[row * N + col] = row + col + 1.0f; 
    }

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start,0 );

    // copy A and B from host to device
    hipMemcpy (dA, A, size, hipMemcpyHostToDevice);
    hipMemcpy (dB, B, size, hipMemcpyHostToDevice);

    // Imvoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);
    MatMul<<<dimGrid,dimBlock>>>(dA, dB, dC);

    // copy C from device  to host 
    hipMemcpy (C, dC, size, hipMemcpyDeviceToHost );

//    for (int row=0; row<N; ++row)  
//    for (int col=0; col<N; ++col) { 
//        printf("%f\n", C[row*N+col]);
//    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float etime;
    hipEventElapsedTime( &etime, start, stop );
    etime = etime/1000.0;

    printf("elapsed time in seconds = %e\n",etime);
    double num_ops = 2.0f*N*N*N;
    printf("number of multiplications and additions  = %e\n",num_ops);
    printf("%e Gflops\n",num_ops/etime/1.e9);

    free(A); free(B); free(C);
    hipFree(dA); hipFree(dB); hipFree(dC);

    return 0;
}
