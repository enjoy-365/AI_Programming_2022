// written by Jongsoo Kim
// Last modification: 2014-06-04
// compile options nvcc -arch sm_20 vector_sum_tb.cu


#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1024*256;

__global__ void add(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;  // handle the data at this index
    while (tid < N) {
       c[tid] = a[tid] + b[tid];
       tid += blockDim.x * gridDim.x;
    }
}

int main (void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the CPU
    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy ( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy ( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );
    
    add<<<256,256>>>(dev_a, dev_b, dev_c);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy ( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );

    // verify that the GPU did the work we requested
    for (int i=0; i<N; i++) {
        if ((a[i]+b[i]) != c[i]) {
        printf("Error: %d + %d != %d", a[i],b[i],c[i]);
        }
    }

    // free the memory allocated on the CPU
    hipFree (dev_a);
    hipFree (dev_b);
    hipFree (dev_c);

    return 0;
}
    
