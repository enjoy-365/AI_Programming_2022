// written by Jongsoo Kim
// last modification: 2014-06-21
// compile options nvcc -arch sm_20 -l cublas dot_product.cu


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

#define sum_squares(x) (x*(x+1)*(2*x+1)/6)

const int N = 2048;

int main (void) { 

    float *a, *b, c;
    float *dev_a, *dev_b;
    hipblasHandle_t handle=0;

    // allocate memory on the CPU side
    a = (float*) malloc ( N*sizeof(float) );
    b = (float*) malloc ( N*sizeof(float) );

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N*sizeof(float) );
    hipMalloc( (void**)&dev_b, N*sizeof(float) );

    for (int i=0; i<N; i++) {
       a[i] = (float) i;
       b[i] = (float) i;
    }

    hipMemcpy ( dev_a, a, N*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy ( dev_b, b, N*sizeof(float), hipMemcpyHostToDevice );

    hipblasCreate(&handle); //cuBlas library initialization
    hipblasSdot(handle, N, dev_a, 1, dev_b, 1, &c);
    
    printf("Dot prodoct of a and b = %f\n", c);
    printf("sum_squares of (N-1) = %f\n", sum_squares((float)(N-1)) );

    hipblasDestroy(handle);
    // free memory on the CUP side
    free(a); free(b);

    // free memory on the GPU side
    hipFree(dev_a); hipFree(dev_b);

    return 0;
}
