// last modification: 2011-06-28
// compile options nvcc dot_product.cu


#include <hip/hip_runtime.h>
#include <stdio.h>

#define imin(a,b) (a<b?a:b)
#define sum_squares(x) (x*(x+1)*(2*x+1)/6)

const int N = 2048;
const int threadsPerBlock = 256;
const int blocksPerGrid = 4; 
//const int blocksPerGrid = N/256; 
//const int blocksPerGrid = (N+threadsPerBlock-1)/threadsPerBlock; 

__global__ void dot_product(float *a, float *b, float *c) { 

    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // set the cache values
    cache[cacheIndex] = temp;

    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock (blockDim.x) must be a power of 2
    // becuase of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex ==0) c[blockIdx.x] = cache[0];

}

int main (void) { 

    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    // allocate memory on the CPU side
    a = (float*) malloc ( N*sizeof(float) );
    b = (float*) malloc ( N*sizeof(float) );
    partial_c = (float*) malloc ( blocksPerGrid*sizeof(float) );

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N*sizeof(float) );
    hipMalloc( (void**)&dev_b, N*sizeof(float) );
    hipMalloc( (void**)&dev_partial_c, blocksPerGrid*sizeof(float) );

    for (int i=0; i<N; i++) {
       a[i] = (float) i;
       b[i] = (float) i;
    }

    hipMemcpy ( dev_a, a, N*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy ( dev_b, b, N*sizeof(float), hipMemcpyHostToDevice );

    dot_product<<<blocksPerGrid,threadsPerBlock>>>(dev_a,dev_b,dev_partial_c);

    hipMemcpy ( partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost );

    c = 0.0f;
    for (int i=0; i<blocksPerGrid; i++)
        c += partial_c[i];

    printf("Dot prodoct of a and b = %f\n", c);
    printf("sum_squares of (N-1) = %f\n", sum_squares((float)(N-1)) );

    // free memory on the CUP side
    free(a); free(b); free(partial_c);

    // free memory on the GPU side
    hipFree(dev_a); hipFree(dev_b); hipFree(dev_partial_c);

    return 0;
}
